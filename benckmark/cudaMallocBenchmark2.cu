#include <hip/hip_runtime.h>
#include <stdio.h>
// main routine
int main() {
  float time;
  hipEvent_t start, stop;

  for (size_t size = 32; size < 1024 * 1024 * 1024; size *= 2) {
    float* d_Data;
    float* h_Data = new float[size];

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_Data, size * sizeof(float));
    hipMemcpy(d_Data, h_Data, size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    if (size > 1024 * 1024) {
      printf("Allocation to device: %fms with size %dMB\n", time,
             static_cast<int>((size * sizeof(float)) / (1024 * 1024)));
    } else if (size > 1024) {
      printf("Allocation to device: %fms with size %dKB\n", time,
             static_cast<int>((size * sizeof(float)) / 1024));
    } else {
      printf("Allocation to device: %fms with size %dB\n", time,
             static_cast<int>(size * sizeof(float)));
    }
    delete[] h_Data;
    hipFree(d_Data);
  }

  return 0;
}